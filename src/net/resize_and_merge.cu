#include "hip/hip_runtime.h"
#include "openposert/gpu/cuda.hpp"
#include "openposert/gpu/cuda_fast_math.hpp"
#include "openposert/net/resize_and_merge.hpp"

namespace openposert {

const auto THREADS_PER_BLOCK_1D = 16u;

template <typename T>
__global__ void fill_kernel(T* target_ptr, const T* const source_ptr,
                            const int n) {
  const auto x = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (x < n) target_ptr[x] = source_ptr[x];
}

template <typename T>
__global__ void resize_kernel(T* target_ptr, const T* const source_ptr,
                              const int width_source, const int height_source,
                              const int width_target, const int height_target) {
  const auto x = (blockIdx.x * blockDim.x) + threadIdx.x;
  const auto y = (blockIdx.y * blockDim.y) + threadIdx.y;
  const auto channel = (blockIdx.z * blockDim.z) + threadIdx.z;
  if (x < width_target && y < height_target) {
    const auto source_area = width_source * height_source;
    const auto target_area = width_target * height_target;
    const T x_source = (x + T(0.5f)) * width_source / T(width_target) - T(0.5f);
    const T y_source =
        (y + T(0.5f)) * height_source / T(height_target) - T(0.5f);
    const T* const source_ptr_channel = source_ptr + channel * source_area;
    target_ptr[channel * target_area + y * width_target + x] =
        bicubic_interpolate(source_ptr_channel, x_source, y_source,
                            width_source, height_source, width_source);
  }
}

template <typename T>
__global__ void resize_and_pad_kernel(T* target_ptr, const T* const source_ptr,
                                      const int width_source,
                                      const int height_source,
                                      const int width_target,
                                      const int height_target,
                                      const T rescale_factor) {
  const auto x = (blockIdx.x * blockDim.x) + threadIdx.x;
  const auto y = (blockIdx.y * blockDim.y) + threadIdx.y;
  const auto channel = (blockIdx.z * blockDim.z) + threadIdx.z;
  if (x < width_target && y < height_target) {
    const auto target_area = width_target * height_target;
    if (x < width_source * rescale_factor &&
        y < height_source * rescale_factor) {
      const auto source_area = width_source * height_source;
      const T x_source = (x + T(0.5f)) / T(rescale_factor) - T(0.5f);
      const T y_source = (y + T(0.5f)) / T(rescale_factor) - T(0.5f);
      const T* const source_ptr_channel = source_ptr + channel * source_area;
      target_ptr[channel * target_area + y * width_target + x] =
          bicubic_interpolate(source_ptr_channel, x_source, y_source,
                              width_source, height_source, width_source);
    } else
      target_ptr[channel * target_area + y * width_target + x] = 0;
  }
}

template <typename T>
__global__ void resize_and_pad_kernel(
    T* target_ptr, const unsigned char* const source_ptr,
    const int width_source, const int height_source, const int width_target,
    const int height_target, const T rescale_factor) {
  const auto x = (blockIdx.x * blockDim.x) + threadIdx.x;
  const auto y = (blockIdx.y * blockDim.y) + threadIdx.y;
  const auto channel = (blockIdx.z * blockDim.z) + threadIdx.z;
  if (x < width_target && y < height_target) {
    const auto target_area = width_target * height_target;
    if (x < width_source * rescale_factor &&
        y < height_source * rescale_factor) {
      const auto source_area = width_source * height_source;
      const T x_source = (x + T(0.5f)) / T(rescale_factor) - T(0.5f);
      const T y_source = (y + T(0.5f)) / T(rescale_factor) - T(0.5f);
      const unsigned char* source_ptr_channel =
          source_ptr + channel * source_area;
      target_ptr[channel * target_area + y * width_target + x] =
          bicubic_interpolate(source_ptr_channel, x_source, y_source,
                              width_source, height_source, width_source);
    } else
      target_ptr[channel * target_area + y * width_target + x] = 0;
  }
}

template <typename T>
void resize_and_pad_rbg_gpu(T* target_ptr, const T* const src_ptr,
                            const int width_source, const int height_source,
                            const int width_target, const int height_target,
                            const T scale_factor) {
  const auto channels = 3;
  const dim3 threads_per_block{THREADS_PER_BLOCK_1D, THREADS_PER_BLOCK_1D, 1};
  const dim3 num_blocks{
      get_number_cuda_blocks(width_target, threads_per_block.x),
      get_number_cuda_blocks(height_target, threads_per_block.y),
      get_number_cuda_blocks(channels, threads_per_block.z)};
  resize_and_pad_kernel<<<num_blocks, threads_per_block>>>(
      target_ptr, src_ptr, width_source, height_source, width_target,
      height_target, scale_factor);
}

template <typename T>
void resize_and_pad_rbg_gpu(T* target_ptr, const unsigned char* const src_ptr,
                            const int width_source, const int height_source,
                            const int width_target, const int height_target,
                            const T scale_factor)

{
  const auto channels = 3;
  const dim3 threads_per_block{THREADS_PER_BLOCK_1D, THREADS_PER_BLOCK_1D, 1};
  const dim3 num_blocks{
      get_number_cuda_blocks(width_target, threads_per_block.x),
      get_number_cuda_blocks(height_target, threads_per_block.y),
      get_number_cuda_blocks(channels, threads_per_block.z)};
  resize_and_pad_kernel<<<num_blocks, threads_per_block>>>(
      target_ptr, src_ptr, width_source, height_source, width_target,
      height_target, scale_factor);
}

template void resize_and_pad_rbg_gpu(
    float* target_ptr, const float* const src_ptr, const int width_source,
    const int height_source, const int width_target, const int height_target,
    const float scale_factor);
template void resize_and_pad_rbg_gpu(
    double* target_ptr, const double* const src_ptr, const int width_source,
    const int height_source, const int width_target, const int height_target,
    const double scale_factor);

template void resize_and_pad_rbg_gpu(
    float* target_ptr, const unsigned char* const src_ptr,
    const int width_source, const int height_source, const int width_target,
    const int height_target, const float scale_factor);
template void resize_and_pad_rbg_gpu(
    double* target_ptr, const unsigned char* const src_ptr,
    const int width_source, const int height_source, const int width_target,
    const int height_target, const double scale_factor);

}  // namespace openposert
